#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>

#define N 512
#define BLOCK_SIZE 16

__global__ void MatAdd(float *A, float *B, 
		       float *C)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // C = A + B (write down your codes)
    if(i < N && j < N)
    {

    }

}

int main()
{

    float *h_A, *h_B, *h_C;
    float *d_A, *d_B, *d_C;

    int i;

    h_A = (float*)malloc(N * N * sizeof(float));
    h_B = (float*)malloc(N * N * sizeof(float));
    h_C = (float*)malloc(N * N * sizeof(float));

    // init data
    for(i = 0; i < (N * N); i++)
    {
	h_A[i] = 1.0;
	h_B[i] = 2.0;
	h_C[i] = 0.0;
    }
    // allocate device memory
    hipMalloc((void**)&d_A,  N * N * sizeof(float));
    hipMalloc((void**)&d_B,  N * N * sizeof(float));
    hipMalloc((void**)&d_C,  N * N * sizeof(float));
    
    // transfer data to device
    hipMemcpy(d_A, h_A, N * N * sizeof(float), hipMemcpyHostToDevice);
    // d_B (write down your codes)

    // d_C (write down your codes)

    

    // fill in correct thread/blocks
    dim3 blockSize( , );
    dim3 numBlock( , );

    // MatAdd kernel
    MatAdd<<<numBlock, blockSize>>>(d_A, d_B, d_C);
    hipDeviceSynchronize();

    // transfer data back to host
    // write down your codes


    for(i = 0; i < (N * N); i++)
    {
    	if(h_C[i]!= 3.0)
	{
	    printf("Error:%f, idx:%d\n", h_C[i], i);
	    break;
	}
    }

    printf("PASS\n");

    // free memory

    free(h_A);
    free(h_B);
    free(h_C);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
